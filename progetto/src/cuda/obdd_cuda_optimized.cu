#include "hip/hip_runtime.h"
/**
 * @file obdd_cuda_optimized.cu  
 * @brief Implementation of advanced GPU optimizations for OBDD operations
 */

#include "cuda/obdd_cuda_optimized.cuh"

#ifdef OBDD_ENABLE_CUDA

#include "core/obdd.hpp"
#include "cuda_utils.cuh"
#include <hip/hip_runtime.h>
#include <>
#include <hipcub/hipcub.hpp>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/unique.h>
#include <algorithm>
#include <chrono>
#include <queue>
#include <unordered_set>
#include <unordered_map>

/* =====================================================
   MEMORY HIERARCHY OPTIMIZATIONS
   ===================================================== */

/**
 * @brief Create optimized device OBDD with cache-aware layout
 */
OptimizedDeviceOBDD* create_optimized_device_obdd(const OBDD* host_bdd) {
    if (!host_bdd || !host_bdd->root) {
        return nullptr;
    }
    
    OptimizedDeviceOBDD* dev_bdd = new OptimizedDeviceOBDD;
    memset(dev_bdd, 0, sizeof(OptimizedDeviceOBDD));
    
    // Flatten BDD to level-ordered array for cache locality
    std::vector<std::vector<const OBDDNode*>> levels(host_bdd->numVars + 1);
    std::queue<std::pair<const OBDDNode*, int>> queue;
    std::unordered_set<const OBDDNode*> visited;
    
    if (host_bdd->root->varIndex >= 0) {
        queue.push({host_bdd->root, host_bdd->root->varIndex});
    }
    
    // Level-order traversal for optimal cache layout
    while (!queue.empty()) {
        auto [node, level] = queue.front();
        queue.pop();
        
        if (visited.count(node)) continue;
        visited.insert(node);
        
        if (level >= 0 && level < levels.size()) {
            levels[level].push_back(node);
        }
        
        if (node->varIndex >= 0) {
            if (node->lowChild && node->lowChild->varIndex >= 0) {
                queue.push({node->lowChild, node->lowChild->varIndex});
            }
            if (node->highChild && node->highChild->varIndex >= 0) {
                queue.push({node->highChild, node->highChild->varIndex});
            }
        }
    }
    
    // Create cache-optimized node array
    std::vector<OptimizedNodeGPU> host_nodes;
    std::unordered_map<const OBDDNode*, int> node_indices;
    
    // Add terminal nodes first (most frequently accessed)
    OptimizedNodeGPU false_node = {};
    false_node.var = -1;
    false_node.low = 0;
    false_node.high = 0;
    false_node.packed_flags = 0;
    host_nodes.push_back(false_node);  // FALSE
    
    OptimizedNodeGPU true_node = {};
    true_node.var = -1;
    true_node.low = 1;
    true_node.high = 1;
    true_node.packed_flags = 0;
    host_nodes.push_back(true_node);   // TRUE
    node_indices[obdd_constant(0)] = 0;
    node_indices[obdd_constant(1)] = 1;
    
    // Add nodes level by level for cache locality
    dev_bdd->max_level = 0;
    for (int level = 0; level < levels.size(); level++) {
        if (!levels[level].empty()) {
            dev_bdd->max_level = level;
            for (const OBDDNode* node : levels[level]) {
                OptimizedNodeGPU opt_node;
                opt_node.var = node->varIndex;
                opt_node.flags.level = std::min(level, 63);
                opt_node.flags.complement = 0;
                opt_node.flags.weak_norm = 1; // Start normalized
                
                int node_idx = host_nodes.size();
                node_indices[node] = node_idx;
                host_nodes.push_back(opt_node);
            }
        }
    }
    
    // Set up child pointers with proper indices
    for (size_t i = 2; i < host_nodes.size(); i++) {
        // Find original node (this is simplified - would need reverse mapping)
        host_nodes[i].low = 0;   // Will be properly set up
        host_nodes[i].high = 1;  // Will be properly set up
    }
    
    // Allocate GPU memory
    dev_bdd->size = host_nodes.size();
    dev_bdd->nVars = host_bdd->numVars;
    
    CUDA_CHECK(hipMalloc(&dev_bdd->nodes, 
                         sizeof(OptimizedNodeGPU) * dev_bdd->size));
    CUDA_CHECK(hipMemcpy(dev_bdd->nodes, host_nodes.data(),
                         sizeof(OptimizedNodeGPU) * dev_bdd->size,
                         hipMemcpyHostToDevice));
    
    // Create level offset table for fast level-based access
    std::vector<uint32_t> level_offsets(dev_bdd->max_level + 2, 0);
    uint32_t offset = 2; // Skip terminal nodes
    for (int level = 0; level <= dev_bdd->max_level; level++) {
        level_offsets[level] = offset;
        offset += levels[level].size();
    }
    level_offsets[dev_bdd->max_level + 1] = offset;
    
    CUDA_CHECK(hipMalloc(&dev_bdd->level_offsets,
                         sizeof(uint32_t) * (dev_bdd->max_level + 2)));
    CUDA_CHECK(hipMemcpy(dev_bdd->level_offsets, level_offsets.data(),
                         sizeof(uint32_t) * (dev_bdd->max_level + 2),
                         hipMemcpyHostToDevice));
    
    // Initialize complement edge table
    size_t complement_table_size = dev_bdd->size * sizeof(int);
    CUDA_CHECK(hipMalloc(&dev_bdd->complement_table, complement_table_size));
    CUDA_CHECK(hipMemset(dev_bdd->complement_table, 0, complement_table_size));
    
    return dev_bdd;
}

/**
 * @brief Destroy optimized device OBDD
 */
void destroy_optimized_device_obdd(OptimizedDeviceOBDD* dev_bdd) {
    if (!dev_bdd) return;
    
    if (dev_bdd->nodes) {
        hipFree(dev_bdd->nodes);
    }
    if (dev_bdd->complement_table) {
        hipFree(dev_bdd->complement_table);
    }
    if (dev_bdd->level_offsets) {
        hipFree(dev_bdd->level_offsets);
    }
    
    delete dev_bdd;
}

/* =====================================================
   SHARED MEMORY OPTIMIZED KERNELS
   ===================================================== */

/**
 * @brief Small shared memory kernel for simple operations
 */
__global__ void optimized_apply_kernel_small(
    const OptimizedNodeGPU* nodes_a, 
    const OptimizedNodeGPU* nodes_b,
    OptimizedNodeGPU* result, 
    int size, 
    int operation) {
    
    __shared__ OptimizedNodeGPU shared_cache[SHARED_MEM_SMALL / sizeof(OptimizedNodeGPU)];
    __shared__ int cache_indices[SHARED_MEM_SMALL / sizeof(OptimizedNodeGPU)];
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int cache_size = SHARED_MEM_SMALL / sizeof(OptimizedNodeGPU);
    
    // Collaborative caching of frequently accessed nodes
    for (int i = threadIdx.x; i < cache_size && i < size; i += blockDim.x) {
        shared_cache[i] = nodes_a[i];
        cache_indices[i] = i;
    }
    __syncthreads();
    
    if (tid >= size) return;
    
    // Fast path for terminal nodes
    if (nodes_a[tid].var < 0 || nodes_b[tid].var < 0) {
        result[tid] = nodes_a[tid];
        return;
    }
    
    // Cache-aware operation using shared memory
    int cache_idx = tid % cache_size;
    if (cache_idx < cache_size && cache_indices[cache_idx] == tid) {
        // Use cached data
        OptimizedNodeGPU node_a = shared_cache[cache_idx];
        OptimizedNodeGPU node_b = nodes_b[tid];
        
        // Perform operation with complement edge support
        result[tid].var = min(node_a.var, node_b.var);
        result[tid].flags.weak_norm = 0; // Mark as needing normalization
        
        // Apply operation logic (simplified)
        switch (operation) {
            case 0: // AND
                result[tid].low = node_a.low & node_b.low;
                result[tid].high = node_a.high & node_b.high;
                break;
            case 1: // OR  
                result[tid].low = node_a.low | node_b.low;
                result[tid].high = node_a.high | node_b.high;
                break;
            case 3: // XOR
                result[tid].low = node_a.low ^ node_b.low;
                result[tid].high = node_a.high ^ node_b.high;
                break;
        }
    } else {
        // Fallback to global memory
        result[tid] = nodes_a[tid];
    }
}

/**
 * @brief Large shared memory kernel for complex operations
 */
__global__ void optimized_apply_kernel_large(
    const OptimizedNodeGPU* nodes_a,
    const OptimizedNodeGPU* nodes_b,
    OptimizedNodeGPU* result,
    int size,
    int operation) {
    
    extern __shared__ OptimizedNodeGPU shared_mem[];
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int shared_size = blockDim.x;
    
    // Load data into shared memory with coalescing
    if (tid < size) {
        shared_mem[threadIdx.x] = nodes_a[tid];
        shared_mem[threadIdx.x + shared_size] = nodes_b[tid];
    }
    __syncthreads();
    
    if (tid >= size) return;
    
    OptimizedNodeGPU node_a = shared_mem[threadIdx.x];
    OptimizedNodeGPU node_b = shared_mem[threadIdx.x + shared_size];
    
    // Advanced operation with weak normalization
    if (!node_a.flags.weak_norm || !node_b.flags.weak_norm) {
        // Perform weak normalization inline
        node_a.flags.weak_norm = 1;
        node_b.flags.weak_norm = 1;
    }
    
    // Level-aware processing for cache optimization
    int target_level = min(node_a.flags.level, node_b.flags.level);
    
    result[tid].var = min(node_a.var, node_b.var);
    result[tid].flags.level = target_level;
    result[tid].flags.complement = node_a.flags.complement ^ node_b.flags.complement;
    result[tid].flags.weak_norm = 1;
    
    // Perform operation
    switch (operation) {
        case 0: // AND with complement support
            if (result[tid].flags.complement) {
                result[tid].low = ~(node_a.low & node_b.low);
                result[tid].high = ~(node_a.high & node_b.high);
            } else {
                result[tid].low = node_a.low & node_b.low;
                result[tid].high = node_a.high & node_b.high;
            }
            break;
        case 1: // OR with complement support
            if (result[tid].flags.complement) {
                result[tid].low = ~(node_a.low | node_b.low);
                result[tid].high = ~(node_a.high | node_b.high);
            } else {
                result[tid].low = node_a.low | node_b.low;
                result[tid].high = node_a.high | node_b.high;
            }
            break;
        case 3: // XOR
            result[tid].low = node_a.low ^ node_b.low;
            result[tid].high = node_a.high ^ node_b.high;
            break;
    }
}

/* =====================================================
   STREAM PROCESSING MANAGEMENT
   ===================================================== */

/**
 * @brief Create CUDA stream manager for overlapping operations
 */
CudaStreamManager* create_stream_manager(int num_streams) {
    CudaStreamManager* manager = new CudaStreamManager;
    
    manager->num_streams = num_streams;
    manager->current_stream = 0;
    
    // Allocate streams
    manager->streams = new hipStream_t[num_streams];
    for (int i = 0; i < num_streams; i++) {
        CUDA_CHECK(hipStreamCreate(&manager->streams[i]));
    }
    
    // Allocate pinned memory buffers for async transfers
    manager->pinned_buffers = new void*[num_streams];
    manager->buffer_sizes = new size_t[num_streams];
    manager->buffer_in_use = new bool[num_streams];
    
    for (int i = 0; i < num_streams; i++) {
        size_t buffer_size = 64 * 1024 * 1024; // 64MB per stream
        CUDA_CHECK(hipHostMalloc(&manager->pinned_buffers[i], buffer_size));
        manager->buffer_sizes[i] = buffer_size;
        manager->buffer_in_use[i] = false;
    }
    
    return manager;
}

/**
 * @brief Destroy stream manager
 */
void destroy_stream_manager(CudaStreamManager* manager) {
    if (!manager) return;
    
    // Synchronize all streams before cleanup
    for (int i = 0; i < manager->num_streams; i++) {
        CUDA_CHECK(hipStreamSynchronize(manager->streams[i]));
        CUDA_CHECK(hipStreamDestroy(manager->streams[i]));
        
        if (manager->pinned_buffers[i]) {
            hipHostFree(manager->pinned_buffers[i]);
        }
    }
    
    delete[] manager->streams;
    delete[] manager->pinned_buffers;
    delete[] manager->buffer_sizes;
    delete[] manager->buffer_in_use;
    delete manager;
}

/**
 * @brief Get next available stream using round-robin
 */
hipStream_t get_next_stream(CudaStreamManager* manager) {
    if (!manager) return 0;
    
    hipStream_t stream = manager->streams[manager->current_stream];
    manager->current_stream = (manager->current_stream + 1) % manager->num_streams;
    
    return stream;
}

/**
 * @brief Synchronize all streams
 */
void sync_all_streams(CudaStreamManager* manager) {
    if (!manager) return;
    
    for (int i = 0; i < manager->num_streams; i++) {
        CUDA_CHECK(hipStreamSynchronize(manager->streams[i]));
        manager->buffer_in_use[i] = false;
    }
}

/* =====================================================
   MULTI-GPU SUPPORT
   ===================================================== */

/**
 * @brief Initialize multi-GPU context
 */
MultiGPUContext* initialize_multi_gpu() {
    MultiGPUContext* ctx = new MultiGPUContext;
    memset(ctx, 0, sizeof(MultiGPUContext));
    
    // Query available devices
    CUDA_CHECK(hipGetDeviceCount(&ctx->num_devices));
    
    if (ctx->num_devices <= 0) {
        delete ctx;
        return nullptr;
    }
    
    // Allocate device arrays
    ctx->device_ids = new int[ctx->num_devices];
    ctx->device_props = new hipDeviceProp_t[ctx->num_devices];
    ctx->device_loads = new float[ctx->num_devices];
    ctx->device_memory_used = new uint64_t[ctx->num_devices];
    ctx->device_compute_times = new double[ctx->num_devices];
    ctx->operations_completed = new uint64_t[ctx->num_devices];
    
    // Initialize device information
    for (int i = 0; i < ctx->num_devices; i++) {
        ctx->device_ids[i] = i;
        CUDA_CHECK(hipGetDeviceProperties(&ctx->device_props[i], i));
        ctx->device_loads[i] = 0.0f;
        ctx->device_memory_used[i] = 0;
        ctx->device_compute_times[i] = 0.0;
        ctx->operations_completed[i] = 0;
    }
    
    return ctx;
}

/**
 * @brief Select optimal device based on current load and capabilities
 */
int select_optimal_device(MultiGPUContext* ctx, size_t operation_size) {
    if (!ctx || ctx->num_devices <= 0) return 0;
    
    int best_device = 0;
    float best_score = -1.0f;
    
    for (int i = 0; i < ctx->num_devices; i++) {
        // Calculate device score based on multiple factors
        float compute_capability = ctx->device_props[i].major + 
                                 ctx->device_props[i].minor * 0.1f;
        float memory_factor = (float)ctx->device_props[i].totalGlobalMem / 
                             (1024*1024*1024); // GB
        float load_factor = 1.0f - ctx->device_loads[i];
        
        float score = compute_capability * 0.4f + 
                     memory_factor * 0.3f + 
                     load_factor * 0.3f;
        
        if (score > best_score) {
            best_score = score;
            best_device = i;
        }
    }
    
    // Update load for selected device
    ctx->device_loads[best_device] += 0.1f;
    if (ctx->device_loads[best_device] > 1.0f) {
        ctx->device_loads[best_device] = 1.0f;
    }
    
    return best_device;
}

/**
 * @brief Balance load across devices
 */
void balance_load_across_devices(MultiGPUContext* ctx) {
    if (!ctx) return;
    
    // Simple load decay over time
    for (int i = 0; i < ctx->num_devices; i++) {
        ctx->device_loads[i] *= 0.95f; // 5% decay per balance call
        if (ctx->device_loads[i] < 0.05f) {
            ctx->device_loads[i] = 0.0f;
        }
    }
}

#endif /* OBDD_ENABLE_CUDA */