#include "hip/hip_runtime.h"
/**
 * @file obdd_cuda.cu
 * @brief Backend GPU (CUDA) per OBDD/ROBDD.
 *
 *  – Copia ("flatten") di un BDD host → vettore compatto di nodi NodeGPU.
 *  – Kernel Breadth-First (BFS) per le operazioni logiche AND / OR / XOR.
 *  – Kernel specializzato NOT (un solo BDD in ingresso).
 *  – Kernel odd–even transposition sort (bubble-sort GPU) per l’ordinamento del
 *    vettore varOrder.
 *  – Wrapper C-linkage: copy, AND, OR, XOR, NOT, var_ordering, free.
 *
 *  Il grafo risultante viene ora ridotto in una ROBDD canonica copiandolo su
 *  host e invocando obdd_reduce().
 */

#include "obdd_cuda.hpp"

#ifdef OBDD_ENABLE_CUDA

#include "obdd.hpp"
#include "obdd_cuda_types.cuh"
#include "cuda_utils.cuh"

#include <hip/hip_runtime.h>
#include <vector>
#include <queue>
#include <unordered_map>
#include <climits>
#include <cstdlib>

/* -------------------------------------------------------------------------- */
/*                     HOST → DEVICE  (flatten + copy)                        */
/* -------------------------------------------------------------------------- */

static void flatten_host(const OBDD* bdd, std::vector<NodeGPU>& out)
{
    out.clear();
    if (!bdd || !bdd->root) {
        out.resize(2);
        out[0] = { -1, 0, 0 };    // FALSE
        out[1] = { -1, 1, 1 };    // TRUE
        return;
    }

    out.reserve(1024);
    out.push_back({ -1, 0, 0 }); // 0 = FALSE
    out.push_back({ -1, 1, 1 }); // 1 = TRUE

    std::queue<const OBDDNode*> Q;
    std::unordered_map<const OBDDNode*, int> idx;

    Q.push(bdd->root);
    while (!Q.empty()) {
        const OBDDNode* cur = Q.front(); Q.pop();
        if (idx.count(cur)) continue;

        int id = static_cast<int>(out.size());
        idx[cur] = id;

        NodeGPU n;
        n.var = cur->varIndex;
        if (cur->varIndex < 0) {
            int v = (cur == obdd_constant(1)) ? 1 : 0;
            n.low = n.high = v;
        } else {
            Q.push(cur->lowChild);
            Q.push(cur->highChild);
            n.low  = -1;
            n.high = -1;
        }
        out.push_back(n);
    }

    for (auto& kv : idx) {
        const OBDDNode* node = kv.first;
        int id               = kv.second;
        if (node->varIndex < 0) continue;
        out[id].low  = (node->lowChild  == obdd_constant(1) || node->lowChild  == obdd_constant(0))
                       ? (node->lowChild == obdd_constant(1) ? 1 : 0)
                       : idx[node->lowChild];
        out[id].high = (node->highChild == obdd_constant(1) || node->highChild == obdd_constant(0))
                       ? (node->highChild == obdd_constant(1) ? 1 : 0)
                       : idx[node->highChild];
    }
}

static DeviceOBDD* copy_flat_to_device(const OBDD* bdd)
{
    std::vector<NodeGPU> host;
    flatten_host(bdd, host);

    DeviceOBDD hostDev{};
    hostDev.size  = static_cast<int>(host.size());
    hostDev.nVars = bdd ? bdd->numVars : 0;

    CUDA_CHECK(hipMalloc(&hostDev.nodes, sizeof(NodeGPU) * hostDev.size));
    CUDA_CHECK(hipMemcpy(hostDev.nodes, host.data(),
                          sizeof(NodeGPU) * hostDev.size,
                          hipMemcpyHostToDevice));

    DeviceOBDD* dHandle = nullptr;
    CUDA_CHECK(hipMalloc(&dHandle, sizeof(DeviceOBDD)));
    CUDA_CHECK(hipMemcpy(dHandle, &hostDev, sizeof(DeviceOBDD), hipMemcpyHostToDevice));
    return dHandle;
}

/* -------------------------------------------------------------------------- */
/*                                KERNELS                                     */
/* -------------------------------------------------------------------------- */

struct Pair { int u, v; };

template<int OP>
__device__ __forceinline__
int logic_op_bit(int a, int b)
{
    if constexpr (OP == 0) return a & b;   // AND
    if constexpr (OP == 1) return a | b;   // OR
    if constexpr (OP == 3) return a ^ b;   // XOR
    return 0;
}

template<int OP>
__global__ void apply_bfs_kernel(const NodeGPU* __restrict__ A,
                                 const NodeGPU* __restrict__ B,
                                 Pair* frontierIn,
                                 int   frontierSize,
                                 Pair* frontierOut,
                                 int*  nextCounter,
                                 NodeGPU* outNodes,
                                 int*  nodeCounter)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= frontierSize) return;

    Pair cur = frontierIn[tid];
    int u = cur.u;
    int v = cur.v;

    if (A[u].var < 0 && B[v].var < 0) {
        int res = logic_op_bit<OP>(A[u].low, B[v].low);
        frontierIn[tid].u = frontierIn[tid].v = res;
        return;
    }

    int varU = (A[u].var < 0) ? INT_MAX : A[u].var;
    int varV = (B[v].var < 0) ? INT_MAX : B[v].var;
    int top  = (varU < varV) ? varU : varV;

    int uLow  = (varU==top) ? A[u].low  : u;
    int uHigh = (varU==top) ? A[u].high : u;
    int vLow  = (varV==top) ? B[v].low  : v;
    int vHigh = (varV==top) ? B[v].high : v;

    int pos = atomicAdd(nextCounter, 2);
    frontierOut[pos]   = { uLow,  vLow  };
    frontierOut[pos+1] = { uHigh, vHigh };

    int myIdx = atomicAdd(nodeCounter, 1);
    outNodes[myIdx] = { top, -1, -1 };

    frontierIn[tid].u = frontierIn[tid].v = myIdx;
}

__global__ void not_kernel(const NodeGPU* inNodes,
                           int*  frontierCur,
                           int   frontierSize,
                           int*  frontierNext,
                           int*  nextCounter,
                           NodeGPU* outNodes,
                           int*  nodeCounter)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= frontierSize) return;

    int u = frontierCur[tid];

    if (inNodes[u].var < 0) {
        frontierCur[tid] = !inNodes[u].low;
        return;
    }

    int var   = inNodes[u].var;
    int uLow  = inNodes[u].low;
    int uHigh = inNodes[u].high;

    int pos = atomicAdd(nextCounter, 2);
    frontierNext[pos]   = uLow;
    frontierNext[pos+1] = uHigh;

    int myIdx = atomicAdd(nodeCounter, 1);
    outNodes[myIdx] = { var, -1, -1 };

    frontierCur[tid] = myIdx;
}

__global__ void oets_phase(int* dArr, int n, int phase)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int i   = 2 * tid + phase;
    if (i + 1 >= n) return;

    int a = dArr[i];
    int b = dArr[i + 1];
    if (a > b) { dArr[i] = b; dArr[i + 1] = a; }
}

/* -------------------------------------------------------------------------- */
/*                              HELPERS HOST                                  */
/* -------------------------------------------------------------------------- */

namespace {

static OBDDNode* rebuild_host_bdd(const std::vector<NodeGPU>& nodes,
                                  int idx,
                                  std::vector<OBDDNode*>& cache)
{
    if (idx == 0) return obdd_constant(0);
    if (idx == 1) return obdd_constant(1);
    if (cache[idx]) return cache[idx];
    const NodeGPU& n = nodes[idx];
    OBDDNode* low  = rebuild_host_bdd(nodes, n.low,  cache);
    OBDDNode* high = rebuild_host_bdd(nodes, n.high, cache);
    cache[idx] = obdd_node_create(n.var, low, high);
    return cache[idx];
}

static void reduce_device_obdd(void** dHandle)
{
    if (!dHandle || !*dHandle) return;

    DeviceOBDD dev{};
    CUDA_CHECK(hipMemcpy(&dev, *dHandle, sizeof(DeviceOBDD), hipMemcpyDeviceToHost));

    std::vector<NodeGPU> nodes(dev.size);
    CUDA_CHECK(hipMemcpy(nodes.data(), dev.nodes,
                          sizeof(NodeGPU) * dev.size,
                          hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(dev.nodes));
    CUDA_CHECK(hipFree(*dHandle));

    std::vector<OBDDNode*> cache(dev.size, nullptr);
    int rootIdx = (dev.size > 2) ? 2 : 0;
    OBDDNode* root = rebuild_host_bdd(nodes, rootIdx, cache);
    OBDDNode* reduced = obdd_reduce(root);

    OBDD tmpUn{root, dev.nVars, static_cast<int*>(std::malloc(sizeof(int)*dev.nVars))};
    obdd_destroy(&tmpUn);

    OBDD tmpRed{reduced, dev.nVars, static_cast<int*>(std::malloc(sizeof(int)*dev.nVars))};
    DeviceOBDD* newDev = copy_flat_to_device(&tmpRed);
    obdd_destroy(&tmpRed);

    *dHandle = static_cast<void*>(newDev);
}

template<int OP>
void gpu_binary_apply(void* dA, void* dB, void** dOut)
{
    DeviceOBDD A{}, B{};
    CUDA_CHECK(hipMemcpy(&A, dA, sizeof(DeviceOBDD), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(&B, dB, sizeof(DeviceOBDD), hipMemcpyDeviceToHost));

    const int MAX_PAIRS = A.size * B.size;
    const int MAX_NODES = 2 + MAX_PAIRS * 2;

    NodeGPU* dNodes = nullptr;
    CUDA_CHECK(hipMalloc(&dNodes, sizeof(NodeGPU) * MAX_NODES));

    const NodeGPU term0 = { -1, 0, 0 };
    const NodeGPU term1 = { -1, 1, 1 };
    CUDA_CHECK(hipMemcpy(dNodes,     &term0, sizeof(NodeGPU), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dNodes + 1, &term1, sizeof(NodeGPU), hipMemcpyHostToDevice));

    Pair *dCur = nullptr, *dNext = nullptr;
    CUDA_CHECK(hipMalloc(&dCur,  sizeof(Pair) * MAX_PAIRS));
    CUDA_CHECK(hipMalloc(&dNext, sizeof(Pair) * MAX_PAIRS));

    Pair start = { 2, 2 };
    CUDA_CHECK(hipMemcpy(dCur, &start, sizeof(Pair), hipMemcpyHostToDevice));

    int *dCurSz=nullptr, *dNextSz=nullptr, *dNodeCnt=nullptr;
    CUDA_CHECK(hipMalloc(&dCurSz,  sizeof(int)));
    CUDA_CHECK(hipMalloc(&dNextSz, sizeof(int)));
    CUDA_CHECK(hipMalloc(&dNodeCnt,sizeof(int)));

    int one = 1, two = 2;
    CUDA_CHECK(hipMemcpy(dCurSz,  &one, sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dNodeCnt,&two, sizeof(int), hipMemcpyHostToDevice));

    while (true) {
        int hFront = 0;
        CUDA_CHECK(hipMemcpy(&hFront, dCurSz, sizeof(int), hipMemcpyDeviceToHost));
        if (hFront == 0) break;

        int blocks = (hFront + OBDD_CUDA_TPB - 1) / OBDD_CUDA_TPB;
        CUDA_CHECK(hipMemset(dNextSz, 0, sizeof(int)));

        apply_bfs_kernel<OP><<<blocks, OBDD_CUDA_TPB>>>(A.nodes, B.nodes,
                                                        dCur, hFront,
                                                        dNext, dNextSz,
                                                        dNodes, dNodeCnt);
        CUDA_CHECK(hipDeviceSynchronize());

        std::swap(dCur, dNext);
        CUDA_CHECK(hipMemcpy(dCurSz, dNextSz, sizeof(int), hipMemcpyDeviceToDevice));
    }

    int hCount = 0;
    CUDA_CHECK(hipMemcpy(&hCount, dNodeCnt, sizeof(int), hipMemcpyDeviceToHost));

    NodeGPU* dCompact = nullptr;
    CUDA_CHECK(hipMalloc(&dCompact, sizeof(NodeGPU) * hCount));
    CUDA_CHECK(hipMemcpy(dCompact, dNodes, sizeof(NodeGPU) * hCount,
                          hipMemcpyDeviceToDevice));

    DeviceOBDD res{ dCompact, hCount, A.nVars };
    DeviceOBDD* dRes = nullptr;
    CUDA_CHECK(hipMalloc(&dRes, sizeof(DeviceOBDD)));
    CUDA_CHECK(hipMemcpy(dRes, &res, sizeof(DeviceOBDD), hipMemcpyHostToDevice));

    *dOut = static_cast<void*>(dRes);

    CUDA_CHECK(hipFree(dNodes));
    CUDA_CHECK(hipFree(dCur));
    CUDA_CHECK(hipFree(dNext));
    CUDA_CHECK(hipFree(dCurSz));
    CUDA_CHECK(hipFree(dNextSz));
    CUDA_CHECK(hipFree(dNodeCnt));

    reduce_device_obdd(dOut);
}

} // anon

/* -------------------------------------------------------------------------- */
/*                              API PUBBLICHE                                 */
/* -------------------------------------------------------------------------- */

extern "C" {

void* obdd_cuda_copy_to_device(const OBDD* bdd)
{
    return static_cast<void*>(copy_flat_to_device(bdd));
}

void obdd_cuda_free_device(void* dHandle)
{
    if (!dHandle) return;
    DeviceOBDD tmp{};
    CUDA_CHECK(hipMemcpy(&tmp, dHandle, sizeof(DeviceOBDD), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(tmp.nodes));
    CUDA_CHECK(hipFree(dHandle));
}

void obdd_cuda_and(void* dA, void* dB, void** dOut)
{
    gpu_binary_apply<0>(dA, dB, dOut);
}

void obdd_cuda_or(void* dA, void* dB, void** dOut)
{
    gpu_binary_apply<1>(dA, dB, dOut);
}

void obdd_cuda_xor(void* dA, void* dB, void** dOut)
{
    gpu_binary_apply<3>(dA, dB, dOut);
}

void obdd_cuda_not(void* dA, void** dOut)
{
    DeviceOBDD A{};
    CUDA_CHECK(hipMemcpy(&A, dA, sizeof(DeviceOBDD), hipMemcpyDeviceToHost));

    const int MAX_NODES = 2 + A.size * 2;

    NodeGPU* dNodes = nullptr;
    CUDA_CHECK(hipMalloc(&dNodes, sizeof(NodeGPU) * MAX_NODES));
    const NodeGPU t0 = { -1, 0, 0 }, t1 = { -1, 1, 1 };
    CUDA_CHECK(hipMemcpy(dNodes,     &t0, sizeof(NodeGPU), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dNodes + 1, &t1, sizeof(NodeGPU), hipMemcpyHostToDevice));

    int *dCur=nullptr, *dNext=nullptr;
    CUDA_CHECK(hipMalloc(&dCur,  sizeof(int) * MAX_NODES));
    CUDA_CHECK(hipMalloc(&dNext, sizeof(int) * MAX_NODES));

    int start = 2;
    CUDA_CHECK(hipMemcpy(dCur, &start, sizeof(int), hipMemcpyHostToDevice));

    int *dCurSz=nullptr, *dNextSz=nullptr, *dNodeCnt=nullptr;
    CUDA_CHECK(hipMalloc(&dCurSz,  sizeof(int)));
    CUDA_CHECK(hipMalloc(&dNextSz, sizeof(int)));
    CUDA_CHECK(hipMalloc(&dNodeCnt,sizeof(int)));

    int one = 1, two = 2;
    CUDA_CHECK(hipMemcpy(dCurSz,  &one, sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dNodeCnt,&two, sizeof(int), hipMemcpyHostToDevice));

    while (true) {
        int hFront = 0;
        CUDA_CHECK(hipMemcpy(&hFront, dCurSz, sizeof(int), hipMemcpyDeviceToHost));
        if (hFront == 0) break;

        int blocks = (hFront + OBDD_CUDA_TPB - 1) / OBDD_CUDA_TPB;
        CUDA_CHECK(hipMemset(dNextSz, 0, sizeof(int)));

        not_kernel<<<blocks, OBDD_CUDA_TPB>>>(A.nodes,
                                              dCur, hFront,
                                              dNext, dNextSz,
                                              dNodes, dNodeCnt);
        CUDA_CHECK(hipDeviceSynchronize());

        std::swap(dCur, dNext);
        CUDA_CHECK(hipMemcpy(dCurSz, dNextSz, sizeof(int), hipMemcpyDeviceToDevice));
    }

    int hCount = 0;
    CUDA_CHECK(hipMemcpy(&hCount, dNodeCnt, sizeof(int), hipMemcpyDeviceToHost));

    NodeGPU* dCompact = nullptr;
    CUDA_CHECK(hipMalloc(&dCompact, sizeof(NodeGPU) * hCount));
    CUDA_CHECK(hipMemcpy(dCompact, dNodes, sizeof(NodeGPU) * hCount,
                          hipMemcpyDeviceToDevice));

    DeviceOBDD res{ dCompact, hCount, A.nVars };
    DeviceOBDD* dRes = nullptr;
    CUDA_CHECK(hipMalloc(&dRes, sizeof(DeviceOBDD)));
    CUDA_CHECK(hipMemcpy(dRes, &res, sizeof(DeviceOBDD), hipMemcpyHostToDevice));

    *dOut = static_cast<void*>(dRes);

    CUDA_CHECK(hipFree(dNodes));
    CUDA_CHECK(hipFree(dCur));
    CUDA_CHECK(hipFree(dNext));
    CUDA_CHECK(hipFree(dCurSz));
    CUDA_CHECK(hipFree(dNextSz));
    CUDA_CHECK(hipFree(dNodeCnt));

    reduce_device_obdd(dOut);
}

void* obdd_cuda_apply(void* dA, void* dB, OBDD_Op op)
{
    void* out = nullptr;
    switch (op) {
        case OBDD_AND: obdd_cuda_and(dA, dB, &out); break;
        case OBDD_OR:  obdd_cuda_or (dA, dB, &out); break;
        case OBDD_XOR: obdd_cuda_xor(dA, dB, &out); break;
        case OBDD_NOT: obdd_cuda_not(dA, &out);     break;
        default: out = nullptr; break;
    }
    return out;
}

void obdd_cuda_var_ordering(int* hostVarOrder, int n)
{
    if (!hostVarOrder || n <= 1) return;

    int* dArr = nullptr;
    CUDA_CHECK(hipMalloc(&dArr, sizeof(int) * n));
    CUDA_CHECK(hipMemcpy(dArr, hostVarOrder, sizeof(int) * n, hipMemcpyHostToDevice));

    int maxPairs = (n + 1) / 2;
    int blocks   = (maxPairs + OBDD_CUDA_TPB - 1) / OBDD_CUDA_TPB;

    for (int pass = 0; pass < n; ++pass) {
        int phase = pass & 1;
        oets_phase<<<blocks, OBDD_CUDA_TPB>>>(dArr, n, phase);
    }
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(hostVarOrder, dArr, sizeof(int) * n, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(dArr));
}

} /* extern "C" */

#endif /* OBDD_ENABLE_CUDA */
